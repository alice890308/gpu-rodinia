
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define size 1024*1024*1024
#define cudaCheckError() {                                          \
    hipError_t e=hipGetLastError();                                 \
    if(e!=hipSuccess) {                                              \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
        exit(0); \
    }                                                                 \
}

int main() {
    float *deviceArr;
    float *hostArr = (float *) malloc(size * sizeof(float));

    hipMalloc((void**) &deviceArr, sizeof(float) * size);

    printf("start copy\n");

    for(int i = 0; i < 1000; i++) {
        hipMemcpy(deviceArr, hostArr, size * sizeof(float), hipMemcpyHostToDevice);
        cudaCheckError();
    }
}